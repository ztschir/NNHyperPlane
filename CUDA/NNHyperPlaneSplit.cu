
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <cmath>
#include <algorithm>
#include <hip/hip_runtime_api.h>

//#define USE_GPU

#define Real float
#define Point Real

struct Node{
  bool isLeaf;
  Node *left;
  Node *right;
  Point *leafPoints;
  int numberOfPoints;
};


void generatePoints(Point * points, int numberOfPoints, int numberOfDimensions){
  //srand (time(NULL));
  srand(0);
  for(int i = 0; i < numberOfPoints; i++)
    for(int j = 0; j < numberOfDimensions; j++)
      points[i*numberOfDimensions + j] = ((double) rand() / (RAND_MAX));
}

/*__global__
void findCentroidDevice(Point * points, Point * center, int numberOfPoints, int numberOfDimensions){
  int i = blockIdx.x;
  if(i < numberOfDimensions){
    Real sum = 0.0;
    //for(int j = 0; j < numberOfPoints; j++){
    //sum += points[j].values[i];
      //}
    sum += points[0].values[i];
    center->values[i] = sum / numberOfPoints;
  }
}
*/
Point * findCentroid(Point * points, int numberOfPoints, int numberOfDimensions){
  Point * center = new Point[numberOfDimensions];
  
  #ifdef USE_GPU
  Point * devicePoints;
  Point * deviceCenter;
  cudaMalloc((void **)&devicePoints, numberOfPoints*sizeof(Point));
  cudaMalloc((void **)&deviceCenter, 1*sizeof(Point));

  cudaMemcpy(devicePoints, points, numberOfPoints*sizeof(Point), cudaMemcpyHostToDevice);
  findCentroidDevice<<<numberOfDimensions, 1>>>(devicePoints, deviceCenter, numberOfPoints, numberOfDimensions);
  cudaMemcpy(center, deviceCenter, 1*sizeof(Point), cudaMemcpyDeviceToHost);

  cudaFree(devicePoints);
  cudaFree(deviceCenter);  
  
  #else
  
  for(int i = 0; i < numberOfDimensions; i++){
    Real sum = 0.0;
    for(int j = 0; j < numberOfPoints; j++)
      sum += points[j*numberOfDimensions + i];
    center[i] = sum / numberOfPoints;
  }
  #endif

  return center;
}

Real findSingleDistance(Point * firstPoint, Point * secondPoint, int numberOfDimensions){
  Real sum = 0.0;
  for(int i = 0; i < numberOfDimensions; i++){
    sum += (firstPoint[i] - secondPoint[i]) * (firstPoint[i] - secondPoint[i]);
  }
  return sqrt(sum);
}

Real * findDistance(Point * firstPoints, Point * diffPoint, int numberOfPoints, int numberOfDimensions){
  Real * distances = new Real[numberOfPoints];
  for(int i = 0; i < numberOfPoints; i++)
    distances[i] = findSingleDistance(&firstPoints[numberOfDimensions*i], diffPoint, numberOfDimensions);
  return distances;
}


int maxPointIndex(Real * distances, int numberOfPoints){
  Real max = 0.0;
  int ind = 0;
  for(int i = 0; i < numberOfPoints; i++){
    if(distances[i] > max){
      max = distances[i];
      ind = i;
    }
  }
  return ind;
}

Point * pointDiff(Point * point1, Point * point2, int numberOfDimensions){
  Point * diff = new Point[numberOfDimensions];
  
  for(int i = 0; i < numberOfDimensions; i++)
    diff[i] = point1[i] - point2[i];

  return diff;
}

Real * projectPoints(Point * points, Point * point2, Point * point1, int numberOfPoints, int numberOfDimensions){
  Real * distance1 = findDistance(points, point1, numberOfPoints, numberOfDimensions);
  Real * distance2 = findDistance(points, point2, numberOfPoints, numberOfDimensions);
  Real * distance = findDistance(point1, point2, 1, numberOfDimensions);

  Real * t = new Real[numberOfPoints];
  for(int i = 0; i < numberOfPoints; i++){
    t[i] = ((distance1[i]*distance1[i] - distance2[i]*distance2[i]) / distance[0]) + distance[0];
    t[i] /= 2.0;
    t[i] /= distance[0];
  }
  return t;
}

template <typename Iterator>
Real median(Iterator begin, Iterator end) {
  Iterator middle = begin + (end - begin) / 2;
   
  std::nth_element(begin, middle, end);
  
  if ((end - begin) % 2 != 0) {
    return *middle;
  } else {
    Iterator lower_middle = std::max_element(begin, middle);
    return (*middle + *lower_middle) / 2.0;
  }
}

Node * hyperplaneSplit(Point * points, int numberOfPoints, int numberOfDimensions, int minNumberOfPoints){
  
  if(numberOfPoints/2 < minNumberOfPoints){
    Node * leaf = new Node();
    leaf->isLeaf = true;
    leaf->leafPoints = points;
    leaf->numberOfPoints = numberOfPoints;
    return leaf;
  }
  
  Node * node = new Node();
  node->isLeaf = false;
  
  Point * center = findCentroid(points, numberOfPoints, numberOfDimensions);
  Real * distance = findDistance(points, center, numberOfPoints, numberOfDimensions);
  Point * Point1 = &points[maxPointIndex(distance, numberOfPoints)*numberOfDimensions];

  distance = findDistance(points, Point1, numberOfPoints, numberOfDimensions);
  Point * Point2 = &points[maxPointIndex(distance, numberOfPoints)*numberOfDimensions];
  Point * diffPoint = pointDiff(Point1, Point2, numberOfDimensions);
  Real * alpha = projectPoints(points, diffPoint, center, numberOfPoints, numberOfDimensions);

  Real * tempAlpha = new Real[numberOfPoints];
  std::copy(alpha, alpha + numberOfPoints, tempAlpha);
  Real mid = median(tempAlpha, tempAlpha + numberOfPoints);

  int numberOfPointsLeft = (int)std::floor((float)numberOfPoints/2.0);
  int numberOfPointsRight = numberOfPoints - numberOfPointsLeft;
  int j = 0;
  Real * temp = new Real[numberOfDimensions];
  for(int i = 0; i < numberOfPointsLeft; i++){
    if(alpha[i] > mid){
      for(j; j < numberOfPointsRight; j++){
        if(alpha[numberOfPointsLeft+j] < mid){
          memcpy(temp, &points[i*numberOfDimensions], numberOfDimensions*sizeof(Real));
          memcpy(&points[i*numberOfDimensions], &points[(numberOfPointsLeft+j)*numberOfDimensions],
                 numberOfDimensions*sizeof(Real));
          memcpy(&points[(numberOfPointsLeft+j)*numberOfDimensions], temp, numberOfDimensions*sizeof(Real));
          j++;
          break;
        }
      }
    }
  }

  node->left = hyperplaneSplit(points, numberOfPointsLeft, numberOfDimensions, minNumberOfPoints);
  node->right = hyperplaneSplit(&points[numberOfPointsLeft*numberOfDimensions], numberOfPointsRight,
                                numberOfDimensions, minNumberOfPoints);
  
  delete temp;
  delete center;
  delete tempAlpha;
  return node;
}


void traverseTree(Node * node, int numberOfDimensions){
  if(!node->isLeaf){
    traverseTree(node->left, numberOfDimensions);    
    traverseTree(node->right, numberOfDimensions);
  }
  else{
    printf("\n");
    for(int i = 0; i < node->numberOfPoints * numberOfDimensions; i += 2){
      printf("%f %f\n", i, node->leafPoints[i], node->leafPoints[i+1]);
    }
  }
}

int main(int argc, char **argv){

  int numberOfPoints = 16;
  int numberOfDimensions = 2;
  
  //int numberOfNeighbors = 4;
  int minNumberOfPoints = 4;
  
  Point points[numberOfPoints*numberOfDimensions];
  generatePoints(points, numberOfPoints, numberOfDimensions);

  Node * topLevelNode = hyperplaneSplit(points, numberOfPoints, numberOfDimensions, minNumberOfPoints);

  traverseTree(topLevelNode, numberOfDimensions);


  hipProfilerStop();
  
  return 0;
}
